#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <algorithm>

using namespace std;

__global__ void merge_sort(float *arr, int length, int chunk)
{
	int start = (blockIdx.x * blockDim.x + threadIdx.x) * chunk;
	if (start >= length)
	{
		return;
	}

	int middle = min(start + chunk / 2, length);
	int end = min(start + chunk, length);
	int left = start;
	int right = middle;
	int i = 0;
	float* temp = new float[chunk]{0.f};

	while (left < middle || right < end)
	{
		float result;
		if (left < middle && right < end)
		{
			result = arr[left] <= arr[right] ? arr[left++] : arr[right++];
		} else 
		{
			result = left < middle ? arr[left++] : arr[right++];
		}
		temp[i++] = result;
	}	

	for (int index = 0; start + index < end; index++)
	{
		arr[start + index] = temp[index];
	}
	delete [] temp;
}

void printArray(float* arr, const int length)
{
	std::stringstream ss;
	ss << "[ ";
	for (int i = 0; i < length; i++)
	{
		ss << arr[i] << ", ";
	}
	std::string str = ss.str();
	str = str.substr(0, str.length() - 2);
	std::cout << str << " ]" << std::endl;
}

int main()
{
	const int length = 100;
	const int size = length * sizeof(float);

	hipError_t cudaStatus;
	hiprandStatus_t hiprandStatus;
	hiprandGenerator_t gen;
	hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

	float* arr;
	cudaStatus = hipMalloc(&arr, size);
	hiprandStatus = hiprandGenerateUniform(gen, arr, length);

	int chunk = 2;
	bool isSorted = false;
	while (!isSorted)
	{
		int threads = ceilf(length / float(chunk));
		int grids = ceilf(threads / 32.f);
		if (grids > 0)
		{
			merge_sort << <grids, 32 >> >(arr, length, chunk);
		} else
		{
			merge_sort << <1, threads >> >(arr, length, chunk);
		}
		if (chunk >= length)
		{
			isSorted = true;
		}
		chunk *= 2;
	}

	float sorted [length];
	hipMemcpy(sorted, arr, size, hipMemcpyDeviceToHost);	
	bool isCorrect = true;
	int i = 0;
	while (isCorrect && i < length - 1)
	{
		isCorrect = sorted[i] <= sorted[i + 1];
		i++;
	}

	cout << "List size: " << length << ", Is correct: " << isCorrect << endl;
	// printArray(sorted, length);
	getchar();
    return 0;
}

