#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include ""
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdio.h>

using namespace std;

__global__ void merge_sort_kernal(float *d_unsorted_arr, float *d_sorted_arr,
                                  uint64_t length, uint64_t chunk) {
  uint64_t start = (blockIdx.x * blockDim.x + threadIdx.x) * chunk;
  if (start >= length) {
    return;
  }

  uint64_t middle = min(start + chunk / 2, length);
  uint64_t end = min(start + chunk, length);
  uint64_t left = start;
  uint64_t right = middle;
  uint64_t index = start;

  while (left < middle || right < end) {
    float result;
    if (left < middle && right < end) {
      result = d_unsorted_arr[left] <= d_unsorted_arr[right]
                   ? d_unsorted_arr[left++]
                   : d_unsorted_arr[right++];
    } else {
      result = left < middle ? d_unsorted_arr[left++] : d_unsorted_arr[right++];
    }
    d_sorted_arr[index++] = result;
  }

  for (uint64_t i = start; i < end; i++) {
    d_unsorted_arr[i] = d_sorted_arr[i];
  }
}

void print_array(float *arr, const uint64_t length) {
  std::stringstream ss;
  ss << "[ ";
  for (uint64_t i = 0; i < length; i++) {
    ss << arr[i] << ", ";
  }
  std::string str = ss.str();
  str = str.substr(0, str.length() - 2);
  std::cout << str << " ]" << std::endl;
}

string timed_operation;
chrono::system_clock::time_point start_time;

void start_timer(string operation) {
  timed_operation = operation;
  start_time = chrono::steady_clock::now();
}

void stop_timer() {
  auto end = chrono::steady_clock::now();
  auto diff = end - start_time;
  cout << timed_operation << " took "
       << chrono::duration<double, milli>(diff).count() << " ms" << endl;
}

void cuda_merge_sort(float *d_unsorted_array, float *d_sorted_array,
                     uint64_t length) {
  uint64_t chunk = 2;
  bool isSorted = false;
  const int threads_per_block = 512;
  // const int threads_per_block = 256;
  // const int threads_per_block = 32;
  while (!isSorted) {
    uint64_t threads = ceilf(length / float(chunk));
    uint64_t grids = ceilf(threads / float(threads_per_block));
    if (grids > 0) {
      merge_sort_kernal<<<grids, threads_per_block>>>(
          d_unsorted_array, d_sorted_array, length, chunk);
    } else {
      merge_sort_kernal<<<1, threads>>>(d_unsorted_array, d_sorted_array,
                                        length, chunk);
    }
    if (chunk >= length) {
      isSorted = true;
    }
    chunk *= 2;
  }
}

void cpu_merge_sort(float *h_unsorted_array, float *h_sorted_array,
                    uint64_t start, uint64_t chunk, uint64_t length) {
  uint64_t middle = min(start + chunk / 2, length);
  uint64_t end = min(start + chunk, length);
  uint64_t left = start;
  uint64_t right = middle;
  uint64_t index = start;

  while (left < middle || right < end) {
    float result;
    if (left < middle && right < end) {
      result = h_unsorted_array[left] <= h_unsorted_array[right]
                   ? h_unsorted_array[left++]
                   : h_unsorted_array[right++];
    } else {
      result =
          left < middle ? h_unsorted_array[left++] : h_unsorted_array[right++];
    }
    h_sorted_array[index++] = result;
  }

  for (uint64_t i = start; i < end; i++) {
    h_unsorted_array[i] = h_sorted_array[i];
  }
}

void cpu_merge_sort(float *h_unsorted_array, float *h_sorted_array,
                    uint64_t length) {
  uint64_t chunk = 2;
  bool isSorted = false;
  while (!isSorted) {
    uint64_t threads = ceilf(length / float(chunk));
    for (uint64_t i = 0; i < threads; i++) {
      cpu_merge_sort(h_unsorted_array, h_sorted_array, i * chunk, chunk,
                     length);
    }
    if (chunk >= length) {
      isSorted = true;
    }
    chunk *= 2;
  }
}

void check_sorted_array(float *sorted_array, uint64_t length) {
  bool isCorrect = true;
  uint64_t i = 0;
  while (isCorrect && i < length - 1) {
    isCorrect = sorted_array[i] <= sorted_array[i + 1];
    i++;
  }
  cout << "List size: " << length
       << ", Is correct: " << (isCorrect ? "Yes" : "No") << endl;
}

// TODO(domenicd): implement a third version in OpenMP and run on Chapman
// numa server to compare CUDA and OpenMP.
int main() {
  // This is the largest we can do without having to do more
  // sofisticated memory management. Becuase this takes up all
  // the avaliable memory on the GPU (6GB).
  // Takes the GPU 2,283 ms and the CPU 323,186 ms.
  // uint64_t length = 1610612736 / 2;

  // A quicker experiment.
  // Takes the GPU 2,161 ms and the CPU 15,630 ms.
  uint64_t length = 10000000 * 5;

  uint64_t size = length * sizeof(float);

  hipError_t cudaStatus;
  hiprandStatus_t hiprandStatus;
  hiprandGenerator_t gen;
  hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
  chrono::system_clock::time_point end;

  float *d_unsorted_array;
  float *d_sorted_array;
  cudaStatus = hipMalloc(&d_unsorted_array, size);
  cudaStatus = hipMalloc(&d_sorted_array, size);

  start_timer("Random number generation");
  hiprandStatus = hiprandGenerateUniform(gen, d_unsorted_array, length);
  hipDeviceSynchronize();
  stop_timer();

  // Store the same sequence of random numbers to use in all tests
  float *h_unsorted_array = new float[length];
  hipMemcpy(h_unsorted_array, d_unsorted_array, size, hipMemcpyDeviceToHost);

  start_timer("CUDA sorting");
  cuda_merge_sort(d_unsorted_array, d_sorted_array, length);
  // Copy from device and check result
  float *h_sorted_array = new float[length];
  hipMemcpy(h_sorted_array, d_unsorted_array, size, hipMemcpyDeviceToHost);
  stop_timer();
  check_sorted_array(h_sorted_array, length);

  // Clean up
  hipFree(d_sorted_array);
  hipFree(d_unsorted_array);
  delete[] h_sorted_array;

  // Reallocate h_sorted_array and perform sorting on CPU
  h_sorted_array = new float[length];
  start_timer("CPU sorting");
  cpu_merge_sort(h_unsorted_array, h_sorted_array, length);
  stop_timer();
  check_sorted_array(h_sorted_array, length);

  // Clean up
  delete[] h_sorted_array;
  delete[] h_unsorted_array;

  return 0;
}
