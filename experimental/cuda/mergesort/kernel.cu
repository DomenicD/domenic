#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <algorithm>

using namespace std;

__global__ void merge_sort(float *arr, long length, long chunk)
{
	long start = (blockIdx.x * blockDim.x + threadIdx.x) * chunk;
	if (start >= length)
	{
		return;
	}

	long middle = min(start + chunk / 2, length);
	long end = min(start + chunk, length);
	long left = start;
	long right = middle;
	long i = 0;
	float* temp = new float[chunk]{0.f};

	while (left < middle || right < end)
	{
		float result;
		if (left < middle && right < end)
		{
			result = arr[left] <= arr[right] ? arr[left++] : arr[right++];
		} else 
		{
			result = left < middle ? arr[left++] : arr[right++];
		}
		temp[i++] = result;
	}	

	for (long index = 0; start + index < end; index++)
	{
		arr[start + index] = temp[index];
	}
	delete [] temp;
}

void printArray(float* arr, const long length)
{
	std::stringstream ss;
	ss << "[ ";
	for (long i = 0; i < length; i++)
	{
		ss << arr[i] << ", ";
	}
	std::string str = ss.str();
	str = str.substr(0, str.length() - 2);
	std::cout << str << " ]" << std::endl;
}

int main()
{
	long length = 2147483640;
	long size = length * sizeof(float);

	hipError_t cudaStatus;
	hiprandStatus_t hiprandStatus;
	hiprandGenerator_t gen;
	hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

	float* arr;
	cudaStatus = hipMalloc(&arr, size);
	hiprandStatus = hiprandGenerateUniform(gen, arr, length);

	long chunk = 2;
	bool isSorted = false;
	while (!isSorted)
	{
		long threads = ceilf(length / float(chunk));
		long grids = ceilf(threads / 32.f);
		if (grids > 0)
		{
			merge_sort << <grids, 32 >> >(arr, length, chunk);
		} else
		{
			merge_sort << <1, threads >> >(arr, length, chunk);
		}
		if (chunk >= length)
		{
			isSorted = true;
		}
		chunk *= 2;
	}

	float* sorted = new float[length]{0.f};
	hipMemcpy(sorted, arr, size, hipMemcpyDeviceToHost);	
	bool isCorrect = true;
	long i = 0;
	while (isCorrect && i < length - 1)
	{
		isCorrect = sorted[i] <= sorted[i + 1];
		i++;
	}	
	cout << "List size: " << length << ", Is correct: " << isCorrect << endl;
	// printArray(sorted, length);
	delete [] sorted;
	getchar();
    return 0;
}

